// Write a parallel CUDA code to solve the one dimensional heat equation using a finite difference scheme for the discretization in space and the Euler method for time integration and validate it
#include <iostream>
#include <hip/hip_runtime.h>

#define NX 100  // Number of grid points in x-direction
#define NT 1  // Number of time steps
#define LX 1.0  // Length of the domain in x-direction
#define T_FINAL 0.1  // Final time
#define ALPHA 0.1  // Thermal diffusivity

#define DT T_FINAL / NT
#define DX LX / (NX -1)

__global__ void heatEquationKernel(float *u, float *u_new, int nx) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx+1 < nx && idx-1>=0) {
    int i = idx;
    float u_xx = (u[i+1] - 2*u[i] + u[i-1]) / (DX*DX);
    u_new[i] = u[i] + ALPHA * DT * u_xx;
  }
}

int main() {
  // Allocate memory on host and device
  float *u, *u_new;
  hipHostMalloc((void **)&u, NX * sizeof(float), hipHostMallocDefault);
  hipHostMalloc((void **)&u_new, NX * sizeof(float), hipHostMallocDefault);
  float *d_u, *d_u_new;
  hipMalloc((void **)&d_u, NX * sizeof(float));
  hipMalloc((void **)&d_u_new, NX * sizeof(float));

  // Initialize the solution array
  for (int i = 0; i < NX; i++) {
    u[i] = sin(M_PI * i * DX);
  }
  hipMemcpy(d_u, u, NX * sizeof(float), hipMemcpyHostToDevice);

  // Set up the kernel launch parameters
  int blockSize = 256;
  int numBlocks = (NX + blockSize - 1) / blockSize;

  // Time-stepping loop
  for (int n = 0; n < NT; n++) {
    heatEquationKernel<<<numBlocks, blockSize>>>(d_u, d_u_new, NX);
    hipDeviceSynchronize();

    // Swap the solution arrays
    float *temp = d_u;
    d_u = d_u_new;
    d_u_new = temp;
  }

  // Copy the solution back to the host
  hipMemcpy(u, d_u, NX * sizeof(float), hipMemcpyDeviceToHost);

  // Validate the solution
  float L2_error = 0.0;
  for (int i = 0; i < NX; i++) {
    float exact = sin(M_PI * i * DX) * exp(-ALPHA * NT * DT * M_PI * M_PI);
    L2_error += (u[i] - exact) * (u[i] - exact);
  }
  L2_error = sqrt(L2_error / NX);
  std::cout << "L2 error: " << L2_error << std::endl;

  // Free memory
  hipHostFree(u);
  hipHostFree(u_new);
  hipFree(d_u);
  hipFree(d_u_new);
  return 0;
}
